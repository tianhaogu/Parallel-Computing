
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

using namespace std;

__global__ void MatrixIteration(double *d_A, double *d_B, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n * n) {
        int row = idx / n, col = idx % n;
        if (row == 0 || row == n - 1 || col == 0 || col == n - 1) {
            d_B[idx] = d_A[idx];
        }
        else {
            double around[4];
            around[0] = d_A[idx+n+1];
            around[1] = d_A[idx+n-1];
            around[2] = d_A[idx-n+1];
            around[3] = d_A[idx-n-1];
            if (around[1] < around[0]) {
                double tmp = around[1];
                around[1] = around[0];
                around[0] = tmp;
            }
            if (around[3] < around[2]) {
                double tmp = around[3];
                around[3] = around[2];
                around[2] = tmp;
            }
            double increment = (around[0] < around[2]) ? (min(around[1], around[2])) : (min(around[0], around[3]));
            d_B[idx] = d_A[idx] + increment;
        }
    }
}

__global__ void CalculateSum(double *d_C, int dist, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n * n && idx % (dist * 2) == 0 && idx + dist < n * n) {
        d_C[idx] += d_C[idx + dist];
    }
}

__global__ void VerifyResult(double *d_F, double *d_A, double *d_C, int n) {
    d_F[0] = d_C[0];
    d_F[1] = d_A[37 * n + 47];
}

int main(int argc, char** argv) {
    int n = atoi(argv[1]), t = atoi(argv[2]);
    int m_size = n * n * sizeof(double);
    double *A = (double*)malloc(m_size);
    for (int i = 0; i < n * n; ++i) {
        int row = i / n, col = i % n;
        A[i] = (1 + cos(2 * row) + sin(col)) * (1 + cos(2 * row) + sin(col));
    }
    int thread_per_block = 1024;
    int block_per_grid = n * n / thread_per_block + 1;
    dim3 gridDim(block_per_grid, 1, 1);
    dim3 blockDim(thread_per_block, 1, 1);
    double *d_A;
    double *d_B;
    hipMalloc(&d_A, m_size);
    hipMalloc(&d_B, m_size);
    hipMemcpy(d_A, A, m_size, hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < t; ++i) {
        if (i % 2 == 0) {
            MatrixIteration<<<gridDim, blockDim>>>(d_A, d_B, n);
        }
        else {
            MatrixIteration<<<gridDim, blockDim>>>(d_B, d_A, n);
        }
        hipDeviceSynchronize();
    }
    double *d_C = d_A;
    double *d_F;
    hipMalloc(&d_F, 2 * sizeof(double));
    for (int dist = 1; dist <= n * n; dist *= 2) {
        CalculateSum<<<gridDim, blockDim>>>(d_C, dist, n);
        hipDeviceSynchronize();
    }
    VerifyResult<<<1, 1>>>(d_F, d_A, d_C, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float total_time = 0.0;
    hipEventElapsedTime(&total_time, start, stop);
    double *h_F = (double*)malloc(2 * sizeof(double));
    hipMemcpy(h_F, d_F, 2 * sizeof(double), hipMemcpyDeviceToHost);
    cout << "Results using " << n << '*' << n << " matrix are: " << endl;
    cout << endl;
    cout << "Sum of the matrix A is: " << h_F[0] << endl;
    cout << "Value of the matrix at A(37, 47): " << h_F[1] << endl;
    cout << "Elapsed time is: " << total_time << endl;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_F);
    free(A);
    free(h_F);
    return 0;
}
